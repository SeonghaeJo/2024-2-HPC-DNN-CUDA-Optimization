#include <mpi.h>

#include <cstdio>
#include <omp.h>

#include "layer.h"
#include "model.h"

#define NUM_GPUS 4

/* [Model Parameters]
 * _w: Weight parameter
 * _b: Bias parameter
 */
Parameter *emb_w[NUM_GPUS];
Parameter *conv0_w[NUM_GPUS], *conv0_b[NUM_GPUS];
Parameter *conv1_w[NUM_GPUS], *conv1_b[NUM_GPUS];
Parameter *conv2_w[NUM_GPUS], *conv2_b[NUM_GPUS];
Parameter *conv3_w[NUM_GPUS], *conv3_b[NUM_GPUS];
Parameter *linear0_w[NUM_GPUS], *linear0_b[NUM_GPUS];
Parameter *linear1_w[NUM_GPUS], *linear1_b[NUM_GPUS];
Parameter *linear2_w[NUM_GPUS], *linear2_b[NUM_GPUS];
Parameter *linear3_w[NUM_GPUS], *linear3_b[NUM_GPUS];

void alloc_and_set_parameters(float *param, size_t param_size) {
  size_t pos[4] = {0};

  #pragma omp parallel for num_threads(NUM_GPUS)
  for (int g = 0; g < NUM_GPUS; g++) {
    CHECK_CUDA(hipSetDevice(g));

    emb_w[g] = new Parameter({21635, 4096}, param + pos[g]);
    pos[g] += 21635 * 4096; 

    conv0_w[g] = new Parameter({1024, 4096, 3}, param + pos[g]);
    pos[g] += 1024 * 4096 * 3; 
    conv0_b[g] = new Parameter({1024}, param + pos[g]);
    pos[g] += 1024;

    conv1_w[g] = new Parameter({1024, 4096, 5}, param + pos[g]);
    pos[g] += 1024 * 4096 * 5; 
    conv1_b[g] = new Parameter({1024}, param + pos[g]);
    pos[g] += 1024;

    conv2_w[g] = new Parameter({1024, 4096, 7}, param + pos[g]);
    pos[g] += 1024 * 4096 * 7;
    conv2_b[g] = new Parameter({1024}, param + pos[g]);
    pos[g] += 1024;

    conv3_w[g] = new Parameter({1024, 4096, 9}, param + pos[g]);
    pos[g] += 1024 * 4096 * 9;
    conv3_b[g] = new Parameter({1024}, param + pos[g]);
    pos[g] += 1024;

    linear0_w[g] = new Parameter({2048, 4096}, param + pos[g]);
    pos[g] += 2048 * 4096;
    linear0_b[g] = new Parameter({2048}, param + pos[g]);
    pos[g] += 2048;

    linear1_w[g] = new Parameter({1024, 2048}, param + pos[g]);
    pos[g] += 1024 * 2048;
    linear1_b[g] = new Parameter({1024}, param + pos[g]);
    pos[g] += 1024;

    linear2_w[g] = new Parameter({512, 1024}, param + pos[g]);
    pos[g] += 512 * 1024;
    linear2_b[g] = new Parameter({512}, param + pos[g]);
    pos[g] += 512;

    linear3_w[g] = new Parameter({2, 512}, param + pos[g]);
    pos[g] += 2 * 512;
    linear3_b[g] = new Parameter({2}, param + pos[g]);
    pos[g] += 2;
  }

  if (pos[0] != param_size) {
    fprintf(stderr, "Parameter size mismatched: %zu != %zu\n", 
            pos[0], param_size);
    exit(EXIT_FAILURE);
  }
}

void free_parameters() {
  for (int g = 0; g < NUM_GPUS; g++) {
    delete emb_w[g];
    delete conv0_w[g];
    delete conv0_b[g];
    delete conv1_w[g];
    delete conv1_b[g];
    delete conv2_w[g];
    delete conv2_b[g];
    delete conv3_w[g];
    delete conv3_b[g];
    delete linear0_w[g];
    delete linear0_b[g];
    delete linear1_w[g];
    delete linear1_b[g];
    delete linear2_w[g];
    delete linear2_b[g];
    delete linear3_w[g];
    delete linear3_b[g];
  }
}

/* [Model Activations] 
 * _a: Activation buffer
 */
Activation *emb_a[NUM_GPUS];
Activation *permute_a[NUM_GPUS];
Activation *conv0_a[NUM_GPUS], *pool0_a[NUM_GPUS];
Activation *conv1_a[NUM_GPUS], *pool1_a[NUM_GPUS];
Activation *conv2_a[NUM_GPUS], *pool2_a[NUM_GPUS];
Activation *conv3_a[NUM_GPUS], *pool3_a[NUM_GPUS];
Activation *concat_a[NUM_GPUS];
Activation *linear0_a[NUM_GPUS], *linear1_a[NUM_GPUS], *linear2_a[NUM_GPUS], *linear3_a[NUM_GPUS];

int *inputs_d[NUM_GPUS];

void alloc_activations() {
  #pragma omp parallel for num_threads(NUM_GPUS)
  for (int g = 0; g < NUM_GPUS; g++) {
    emb_a[g] = new Activation({NUM_SENTENCES / NUM_GPUS, SEQ_LEN, 4096});
    permute_a[g] = new Activation({NUM_SENTENCES / NUM_GPUS, 4096, SEQ_LEN});
    conv0_a[g] = new Activation({NUM_SENTENCES / NUM_GPUS, 1024, SEQ_LEN - 2});
    pool0_a[g] = new Activation({NUM_SENTENCES / NUM_GPUS, 1024});
    conv1_a[g] = new Activation({NUM_SENTENCES / NUM_GPUS, 1024, SEQ_LEN - 4});
    pool1_a[g] = new Activation({NUM_SENTENCES / NUM_GPUS, 1024});
    conv2_a[g] = new Activation({NUM_SENTENCES / NUM_GPUS, 1024, SEQ_LEN - 6});
    pool2_a[g] = new Activation({NUM_SENTENCES / NUM_GPUS, 1024});
    conv3_a[g] = new Activation({NUM_SENTENCES / NUM_GPUS, 1024, SEQ_LEN - 8});
    pool3_a[g] = new Activation({NUM_SENTENCES / NUM_GPUS, 1024});
    concat_a[g] = new Activation({NUM_SENTENCES / NUM_GPUS, 4096});
    linear0_a[g] = new Activation({NUM_SENTENCES / NUM_GPUS, 2048});
    linear1_a[g] = new Activation({NUM_SENTENCES / NUM_GPUS, 1024});
    linear2_a[g] = new Activation({NUM_SENTENCES / NUM_GPUS, 512});
    linear3_a[g] = new Activation({NUM_SENTENCES / NUM_GPUS, 2});

    CHECK_CUDA(hipMalloc(&inputs_d[g], NUM_SENTENCES / NUM_GPUS * SEQ_LEN * sizeof(int)));
  }
}

void free_activations() {
  for (int g = 0; g < NUM_GPUS; g++) {
    delete emb_a[g];
    delete permute_a[g];
    delete conv0_a[g];
    delete pool0_a[g];
    delete conv1_a[g];
    delete pool1_a[g];
    delete conv2_a[g];
    delete pool2_a[g];
    delete conv3_a[g];
    delete pool3_a[g];
    delete concat_a[g];
    delete linear0_a[g];
    delete linear1_a[g];
    delete linear2_a[g];
    delete linear3_a[g];

    CHECK_CUDA(hipFree(inputs_d[g]));
  }
}

/* [Model Computation: Sentiment Analysis Task] */
void predict_sentiment(int *inputs, float *outputs, size_t n_samples) {

  if (n_samples != NUM_SENTENCES) {
    printf("predict_sentiment : n_samples is not equal to NUM_SENTENCES");
    exit(1);
  }

  int mpi_rank;
  MPI_Comm_rank(MPI_COMM_WORLD, &mpi_rank);
  if (mpi_rank == 0) {

    #pragma omp parallel for num_threads(NUM_GPUS)
    for (int g = 0; g < NUM_GPUS; g++) {

      // inputs = [num_sentences * SEQ_LEN]
      CHECK_CUDA(hipMemcpy(inputs_d[g], &inputs[(NUM_SENTENCES / NUM_GPUS) * SEQ_LEN * g], 
                            NUM_SENTENCES / NUM_GPUS * SEQ_LEN * sizeof(int), 
                            hipMemcpyHostToDevice));

      Embedding(inputs_d[g], emb_w[g], emb_a[g]);

      Permute(emb_a[g], permute_a[g]);

      Conv1D(permute_a[g], conv0_w[g], conv0_b[g], conv0_a[g]);
      ReLU(conv0_a[g]);
      GetMax(conv0_a[g], pool0_a[g]);

      Conv1D(permute_a[g], conv1_w[g], conv1_b[g], conv1_a[g]);
      ReLU(conv1_a[g]);
      GetMax(conv1_a[g], pool1_a[g]);

      Conv1D(permute_a[g], conv2_w[g], conv2_b[g], conv2_a[g]);
      ReLU(conv2_a[g]);
      GetMax(conv2_a[g], pool2_a[g]);

      Conv1D(permute_a[g], conv3_w[g], conv3_b[g], conv3_a[g]);
      ReLU(conv3_a[g]);
      GetMax(conv3_a[g], pool3_a[g]);

      Concat(pool0_a[g], pool1_a[g], pool2_a[g], pool3_a[g], concat_a[g]);

      Linear(concat_a[g], linear0_w[g], linear0_b[g], linear0_a[g]);
      ReLU(linear0_a[g]);

      Linear(linear0_a[g], linear1_w[g], linear1_b[g], linear1_a[g]);
      ReLU(linear1_a[g]);

      Linear(linear1_a[g], linear2_w[g], linear2_b[g], linear2_a[g]);
      ReLU(linear2_a[g]);

      Linear_narrow(linear2_a[g], linear3_w[g], linear3_b[g], linear3_a[g]);

      // outputs = [num_sentences * N_CLASSES]
      CHECK_CUDA(hipMemcpy(&outputs[(NUM_SENTENCES / NUM_GPUS) * 2 * g], linear3_a[g],
                            NUM_SENTENCES / NUM_GPUS * 2 * sizeof(float),
                            hipMemcpyDeviceToHost));
    }
  }
}